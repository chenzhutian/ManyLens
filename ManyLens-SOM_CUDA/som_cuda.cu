#include "hip/hip_runtime.h"
#include "som_cuda.cuh" 

//Initialize CUDA
int InitializeCUDA(void)
{
	int count = 0;
	std::cout << "Start to detecte devices.........\n";
	hipGetDeviceCount(&count);
	if (count == 0)
	{
		std::cerr << "There is no device.\n";
		return 1;
	}

	std::cout << count << " device detected.\n";
	int i;

	for (i = 0; i < count; i++)
	{
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				std::cout << "Device:" << i + 1 << " supports CUDA " << prop.name << "." << prop.major << "." << prop.minor << std::endl;
				break;
			}
		}
	}
	if (i == count)
	{
		std::cerr << "There is no device supporting CUDA 1.x.\n";
		return 1;
	}
	hipSetDevice(i);


	return 2;
}

//Do clean up
int CleanUp()
{
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

template <typename T> int sgn(T val)
{
	return (T(0) < val) - (val < T(0));
}

void somFree(float* pointer)
{
	delete[] pointer;
	pointer = NULL;
}

bool CPUcheckEuclideanResult(const float* d_result,
	const float* d_weights,
	const float* d_input,
	const unsigned int* d_BID,
	unsigned int inputIndex,
	int neuron_number,
	int batch_number,
	int dimension,
	int flag)
{
	float* h_weights = new float[neuron_number*dimension];
	float* h_input = new float[batch_number*dimension];
	float* h_checkresult = new float[neuron_number*batch_number];
	unsigned int* h_checkBID = new unsigned int[batch_number];
	hipMemcpy(h_checkBID, d_BID, batch_number*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(h_checkresult, d_result, neuron_number*batch_number*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_weights, d_weights, neuron_number*dimension*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_input, d_input + inputIndex*dimension, batch_number*dimension*sizeof(float), hipMemcpyDeviceToHost);
	if (flag == -1)
	{
		inputIndex += 40000000;
	}

	/*--------------- check the intermidiateResult of each batch -----------------*/
	std::ofstream fout("../result/h_intermidiateResult" + std::to_string(inputIndex) + ".txt");
	for (int i = 0; i < batch_number; ++i)
	{
		for (int j = 0; j < neuron_number; ++j)
		{
			if (j == (neuron_number - 1))
				fout << h_checkresult[j + i*neuron_number];
			else
				fout << h_checkresult[j + i*neuron_number] << " ";
		}
		if (i != (batch_number - 1))
			fout << std::endl;
	}
	fout.close();

	/*--------------- check the input of each batch -----------------*/
	fout.open("../result/h_input" + std::to_string(inputIndex) + ".txt");
	for (int i = 0; i < batch_number; ++i)
	{
		for (int j = 0; j < dimension; ++j)
		{

			if (j == (dimension - 1))
				fout << h_input[j + i*dimension];
			else
				fout << h_input[j + i*dimension] << " ";
		}
		if (i != (batch_number - 1))
			fout << std::endl;
	}
	fout.close();

	/*--------------- check the result of weights update -----------------*/
	fout.open("../result/weights" + std::to_string(inputIndex) + ".txt");
	for (int i = 0; i < neuron_number; ++i)
	{
		for (int j = 0; j < dimension; j++)
		{
			if (j == (dimension - 1))
				fout << h_weights[j*neuron_number + i];
			else
				fout << h_weights[j*neuron_number + i] << " ";
		}
		if (i != (neuron_number - 1))
			fout << std::endl;
	}
	fout.close();

	/*------------------ Check the BID ----------------*/
	fout.open("../result/bID" + std::to_string(inputIndex) + ".txt");

	for (int j = 0; j < batch_number; j++)
	{
		if (j == (batch_number - 1))
			fout << h_checkBID[j];
		else
			fout << h_checkBID[j] << std::endl;
	}
	fout.close();


	delete[] h_checkresult;
	delete[] h_input;
	delete[] h_weights;
	delete[] h_checkBID;
	h_checkBID = NULL;
	h_checkresult = NULL;
	h_input = NULL;
	h_weights = NULL;
	return true;
}

//Calculate the distance between each neuron and input vector
//this kernel assumes d_weights is column-major          d_weights(dimension, neuron_number)
//this kernel assumes d_input_set is row-major           d_input_set(batch_size, dimension)
//this kernel assumes d_result is column-major           d_result(batch_size,neuron_number)
//this kernel assumes number of threads per threadblock == DIMENSION
//CHKSIZE is the number of d_weights vectors that will be compute per block
__global__ void Calculate_Euclidean_Distance_Kernel(const float *d_weights,
	const float *d_input_set,
	const unsigned input_index_of_this_batch,
	const int batch_size,
	const int neuron_number,
	float *d_result)
{
	__shared__ float shared_input_set[CHKSIZE * DIMENSION];		//enough shared storage for CHKSIZE vectors of d_input_set	
	int bx = blockIdx.x;										//one block per CHKSIZE rows of d_input_set
	int tx = threadIdx.x;
	float result[CHKSIZE];

	int numCHKSIZE = (bx + 1) * CHKSIZE < batch_size ? CHKSIZE : batch_size - bx*CHKSIZE;
#pragma unroll
	for (int i = 0; i < numCHKSIZE; i++)
		shared_input_set[(i * DIMENSION) + tx] = d_input_set[((input_index_of_this_batch + (bx * CHKSIZE) + i) * DIMENSION) + tx];
	__syncthreads();

	//loop across all vectors in d_weights
	while (tx < neuron_number)
	{
#pragma unroll
		for (int i = 0; i < numCHKSIZE; i++)
			result[i] = 0.0f;

		for (int i = 0; i < DIMENSION; i++)
		{
			float Atemp = d_weights[(neuron_number * i) + tx];
			//compute all CHKSIZE d_input_set vectors with read of d_weights
#pragma unroll
			for (int j = 0; j < numCHKSIZE; j++)
			{
				float temp = Atemp - shared_input_set[i + (j * DIMENSION)];
				result[j] += temp * temp;
			}
		}

		//store CHKSIZE results
#pragma unroll
		for (int i = 0; i < numCHKSIZE; i++)
			d_result[(i + (bx * CHKSIZE)) * neuron_number + tx] = result[i];

		tx += blockDim.x;
	}
}


//Has not finish yet
__global__ void Calculate_Cosine_Distance_Kernel(const float *d_weights,
	const float *d_input_set,
	const unsigned input_index_of_this_batch,
	const int batch_size,
	const int neuron_number,
	float *d_result)
{
	__shared__ float shared_input_set[CHKSIZE * DIMENSION];		//enough shared storage for CHKSIZE vectors of d_input_set	
	int bx = blockIdx.x;										//one block per CHKSIZE rows of d_input_set
	int tx = threadIdx.x;
	float result[CHKSIZE];

	int numCHKSIZE = (bx + 1) * CHKSIZE < batch_size ? CHKSIZE : batch_size - bx*CHKSIZE;
#pragma unroll
	for (int i = 0; i < numCHKSIZE; i++)
		shared_input_set[(i * DIMENSION) + tx] = d_input_set[((input_index_of_this_batch + (bx * CHKSIZE) + i) * DIMENSION) + tx];
	__syncthreads();

	//loop across all vectors in d_weights
	while (tx < neuron_number)
	{
#pragma unroll
		for (int i = 0; i < numCHKSIZE; i++)
			result[i] = 0.0f;

		for (int i = 0; i < DIMENSION; i++)
		{
			float Atemp = d_weights[(neuron_number * i) + tx];
			//compute all CHKSIZE d_input_set vectors with read of d_weights
#pragma unroll
			for (int j = 0; j < numCHKSIZE; j++)
			{
				float temp = Atemp - shared_input_set[i + (j * DIMENSION)];
				result[j] += temp * temp;
			}
		}

		//store CHKSIZE results
#pragma unroll
		for (int i = 0; i < numCHKSIZE; i++)
			d_result[(i + (bx * CHKSIZE)) * neuron_number + tx] = result[i];

		tx += blockDim.x;
	}
}

__global__ void Calculate_Euclidean_Distance_Kernel_noRandomMapping(const float *d_weights,
	const int dimension,
	const float *d_input_set,
	const unsigned input_index_of_this_batch,
	const int batch_size,
	const int neuron_number,
	float *d_result)
{
	extern __shared__ float shared_input_set[];		//enough shared storage for CHKSIZE vectors of d_input_set	
	int bx = blockIdx.x;										//one block per CHKSIZE rows of d_input_set
	int tx = threadIdx.x;
	float result = 0.0f;

	while (tx < dimension)
	{
		shared_input_set[tx] = d_input_set[(input_index_of_this_batch + bx) * dimension + tx];
		tx += blockDim.x;
	}
	__syncthreads();
	tx = threadIdx.x;

	//loop across all vectors in d_weights
	while (tx < neuron_number)
	{

		for (int i = 0; i < dimension; i++)
		{
			float Atemp = d_weights[(neuron_number * i) + tx];
			//compute all CHKSIZE d_input_set vectors with read of d_weights
			float temp = Atemp - shared_input_set[i];
			result += temp * temp;
		}

		//store CHKSIZE results								
		d_result[bx * neuron_number + tx] = result;
		tx += blockDim.x;
	}
}

//Find out the index of min element
__global__ void Min_Reduce_Kernel(const float* d_result, unsigned int* d_BID, const size_t neuronNum)
{
	__shared__ float sValue[512];
	__shared__ int sIndex[512];

	int tx = threadIdx.x;
	int stride = blockIdx.x*neuronNum;
	int gid = tx + stride;
	int upper = stride + neuronNum;

	sValue[tx] = FLT_MAX;
	sIndex[tx] = gid;
	float temp;
	while (gid < upper) {
		temp = d_result[gid];

		sIndex[tx] = sValue[tx] > temp ? gid : sIndex[tx];
		sValue[tx] = sValue[tx] > temp ? temp : sValue[tx];

		gid += blockDim.x;
	}
	__syncthreads();

	if (tx < 256)
	{
		sIndex[tx] = sValue[tx] > sValue[tx + 256] ? sIndex[tx + 256] : sIndex[tx];
		sValue[tx] = sValue[tx] > sValue[tx + 256] ? sValue[tx + 256] : sValue[tx];
	}
	__syncthreads();
	if (tx < 128)
	{
		sIndex[tx] = sValue[tx] > sValue[tx + 128] ? sIndex[tx + 128] : sIndex[tx];
		sValue[tx] = sValue[tx] > sValue[tx + 128] ? sValue[tx + 128] : sValue[tx];
	}
	__syncthreads();
	if (tx < 64)
	{
		sIndex[tx] = sValue[tx] > sValue[tx + 64] ? sIndex[tx + 64] : sIndex[tx];
		sValue[tx] = sValue[tx] > sValue[tx + 64] ? sValue[tx + 64] : sValue[tx];
	}
	__syncthreads();
	if (tx < 32)
	{
		if (sValue[tx] > sValue[tx + 32])
		{
			sValue[tx] = sValue[tx + 32];
			sIndex[tx] = sIndex[tx + 32];
		}
		if (sValue[tx] > sValue[tx + 16])
		{
			sValue[tx] = sValue[tx + 16];
			sIndex[tx] = sIndex[tx + 16];
		}
		if (sValue[tx] > sValue[tx + 8])
		{
			sValue[tx] = sValue[tx + 8];
			sIndex[tx] = sIndex[tx + 8];
		}
		if (sValue[tx] > sValue[tx + 4])
		{
			sValue[tx] = sValue[tx + 4];
			sIndex[tx] = sIndex[tx + 4];
		}

		if (sValue[tx] > sValue[tx + 2])
		{
			sValue[tx] = sValue[tx + 2];
			sIndex[tx] = sIndex[tx + 2];
		}
		//sIndex[tx] = sValue[tx] > sValue[tx+2] ? sIndex[tx+2] : sIndex[tx];
		//sValue[tx] = sValue[tx] > sValue[tx+2] ? sValue[tx+2] : sValue[tx];

		sIndex[tx] = sValue[tx] > sValue[tx + 1] ? sIndex[tx + 1] : sIndex[tx];
		sValue[tx] = sValue[tx] > sValue[tx + 1] ? sValue[tx + 1] : sValue[tx];
	}
	if (tx == 0)
	{
		d_BID[blockIdx.x] = sIndex[0] - stride;
		//d_error[blockIdx.x] = sValue[0];
	}
}

//Find the best match neuron by using bathc kernel function.
bool Find_Best_Match_Neuron(const float* d_weights,
	const unsigned int neuron_number,
	const float* d_input_set,
	const unsigned int input_index_of_this_batch,
	const unsigned int batch_size,
	unsigned int* d_BID,
	float* d_result)
{
	hipError_t cudaStatus;
	dim3 threads(DIMENSION);
	dim3 blocks(ceil((double)batch_size / (double)CHKSIZE));
	//Calculate_Euclidean_Distance_Kernel_noRandomMapping<<<batch_size,1024,dimension*sizeof(float)>>>(d_weights,dimension,d_input_set,input_index_of_this_batch,batch_size,neuron_number,d_result);
	Calculate_Euclidean_Distance_Kernel << <blocks, threads >> >(d_weights, d_input_set, input_index_of_this_batch, batch_size, neuron_number, d_result);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching EuclideanDistancesBMU!\n", cudaStatus);
		return false;
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(Min_Reduce_Kernel), hipFuncCachePreferL1);
	Min_Reduce_Kernel << <batch_size, 512 >> >(d_result, d_BID, neuron_number);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching min reduce!\n", cudaStatus);
		return false;
	}

	return true;
}

//Find the best match neuron by using bathc kernel function.
bool output_BID(const float* d_weights,
	const unsigned int neuron_number,
	const float* d_input_set,
	unsigned int input_index_of_this_batch,
	unsigned int batch_size, unsigned int* d_BID, float* d_result)
{
	hipError_t cudaStatus;
	dim3 threads(DIMENSION);
	dim3 blocks(ceil((double)batch_size / (double)CHKSIZE));
	//Calculate_Euclidean_Distance_Kernel_noRandomMapping<<<batch_size,1024,dimension*sizeof(float)>>>(d_weights,dimension,d_input_set,input_index_of_this_batch,batch_size,neuron_number,d_result);
	Calculate_Euclidean_Distance_Kernel << <blocks, threads >> >(d_weights, d_input_set, input_index_of_this_batch, batch_size, neuron_number, d_result);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching EuclideanDistancesBMU!\n", cudaStatus);
		return false;
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(Min_Reduce_Kernel), hipFuncCachePreferL1);
	Min_Reduce_Kernel << <batch_size, 512 >> >(d_result, d_BID, neuron_number);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching min reduce!\n", cudaStatus);
		return false;
	}

	return true;
}

//Update weight of each neuron
__global__ void Bad_Update_Map_Map_Kernel(const float* d_input_set, const int input_index_of_this_batch, const int* d_position, const unsigned int* bID, const unsigned int batch_size, const float fsigmaT, float* d_weights)
{
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	float denominator = 0.f;
	float numerator = 0.f;
	int index_factor = gridDim.x - 1;
	int DIMENSIONxCHKSIZE = DIMENSION*CHKSIZE;
	__shared__ float tempDenominator[DIMENSION*CHKSIZE];		//DIMENSION*CHKSIZE
	int j = 0;
	while (tx < batch_size)
	{

		//Calculate the influence of each input vector
		for (int i = 0; i < CHKSIZE; i++)
		{
			int bid = bID[tx];				//the id of best match neuron
			int tempX = (d_position[2 * bx] - d_position[2 * bid]);
			int tempY = (d_position[2 * bx + 1] - d_position[2 * bid + 1]);
			float tempDist = tempX*tempX + tempY*tempY;
			tempDenominator[tx + (i * DIMENSION)] = bx^bid ? expf(-tempDist / fsigmaT) : 1;
		}
		__syncthreads();

		//Sum up the influence
		for (int i = 0; i < DIMENSIONxCHKSIZE; i++)
		{
			numerator += tempDenominator[i] * d_input_set[threadIdx.x + ((input_index_of_this_batch + (j * DIMENSIONxCHKSIZE) + i) * DIMENSION)];
			denominator += tempDenominator[i];
		}
		++j;
		tx += DIMENSIONxCHKSIZE;
	}
	//Update the weight of each neuron
	d_weights[threadIdx.x * gridDim.x + bx] = denominator;// numerator / 
}

//Update weight of each neuron
__global__ void Update_Map_Map_Kernel(const float* d_input_set,
	const int input_index_of_this_batch,
	const float* d_distance,
	const unsigned int* bID,
	const unsigned int batch_size,
	const float fsigmaT,
	float* d_weights)
{
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	float denominator = 0.f;
	float numerator = 0.f;
	int index_factor = gridDim.x - 1;
	__shared__ float tempDenominator[DIMENSION];		//DIMENSION*CHKSIZE
	int j = 0;
	int count = 0;
	int upper = ceilf((float)batch_size / (float)DIMENSION);
	for (int j = 0; j < upper; ++j)
	{
		if (tx < batch_size)
		{
			int bid = bID[tx];				//the id of best match neuron
			/* Find the bigger one between bid and bx, 'a' is the bigger one*/
			int a = bx + bid;
			int b = bx < bid ? bx : bid;
			a = a - b;
			int index = a + index_factor * b - 1 - (b + 1) * b * 0.5;

			//Calculate the influence of each input vector
			float tempDist = bx^bid ? d_distance[index + 1] : 0;

			tempDenominator[threadIdx.x] = bx^bid ? expf(-tempDist / fsigmaT) : 1;
			__syncthreads();
		}

		count = (j + 1)*DIMENSION < batch_size ? DIMENSION : (batch_size - j * DIMENSION);
		//Sum up the influence
		for (int i = 0; i < count; i++)
		{
			numerator += tempDenominator[i] * d_input_set[threadIdx.x + ((input_index_of_this_batch + (j * DIMENSION) + i) * DIMENSION)];
			denominator += tempDenominator[i];
		}
		tx += DIMENSION;
	}
	//Update the weight of each neuron
	d_weights[threadIdx.x * gridDim.x + bx] = numerator / denominator;
}

__global__ void Update_Map_Map_Kernel_noRandomMapping(const float* d_input_set,
	const int dimension,
	const int input_index_of_this_batch,
	const float* d_distance,
	const unsigned int* bID,
	const unsigned int batch_size,
	const float fsigmaT,
	float* d_weights)
{
	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;
	float denominator = 0.f;
	float numerator = 0.f;
	int index_factor = gridDim.y - 1;
	__shared__ float tempDenominator[512];		//DIMENSION*CHKSIZE
	int j = 0;
	int tid = bx*blockDim.x + threadIdx.x;
	int count = 0;
	while (tx < batch_size)
	{
		int bid = bID[tx];				//the id of best match neuron
		/* Find the bigger one between bid and by, 'a' is the bigger one*/
		int a = by + bid;
		int b = by < bid ? by : bid;
		a = a - b;
		int index = a + index_factor * b - 1 - (b + 1) * b * 0.5;

		//Calculate the influence of each input vector
		float tempDist = by^bid ? d_distance[index + 1] : 0;

		tempDenominator[threadIdx.x] = by^bid ? expf(-tempDist / fsigmaT) : 1;
		__syncthreads();


		if (tid < dimension)
		{
			count = (j + 1) * 512 < batch_size ? 512 : (batch_size - j * 512);
			//Sum up the influence
			for (int i = 0; i < count; i++)
			{
				numerator += tempDenominator[i] * d_input_set[tid + ((input_index_of_this_batch + (j * 512) + i) * dimension)];
				denominator += tempDenominator[i];
			}
		}

		++j;
		tx += blockDim.x;
	}
	if (tid < dimension)
		//Update the weight of each neuron
		d_weights[tid * gridDim.y + by] = numerator / denominator;
}


//Update weight of each neuron vector
bool Update_Map(const float* d_distance,
	const unsigned int neuron_number,
	const float* d_input_set,
	const int input_index_of_this_batch,
	const unsigned int* bID,
	const unsigned int batch_size,
	const int dimension,
	const float fsigmaT,
	float * d_weights)
{

	hipError_t cudaStatus;
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(Update_Map_Map_Kernel), hipFuncCachePreferL1);
	//dim3 blocks(ceil((double)dimension/512.0),neuron_number);
	//Update_Map_Map_Kernel<<<blocks,512>>>(d_input_set,dimension,input_index_of_this_batch,d_distance,bID,batch_size,
	Update_Map_Map_Kernel <<<neuron_number, DIMENSION>>>(d_input_set, input_index_of_this_batch, d_distance, bID, batch_size, fsigmaT, d_weights);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "Update_Map_Map_Kernel returned error code %d after launching Update_Map_Map_Kernel!\n", cudaStatus);
		std::cout << input_index_of_this_batch << " " << batch_size << " " << fsigmaT << std::endl;
		std::ofstream ferr_BID("../result/error_BID.txt");
		std::ofstream ferr_Input("../result/error_input.txt");
		int * e_BID = new int[batch_size];
		float * e_input_set = new float[dimension*batch_size];
		hipMemcpy(e_BID, bID, batch_size*sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(e_input_set, d_input_set + dimension*input_index_of_this_batch, dimension*batch_size*sizeof(float), hipMemcpyDeviceToHost);
		for (int i = 0; i < batch_size; i++)
		{
			ferr_BID << e_BID[i] << std::endl;
			for (int j = 0; j < dimension; ++j)
			{
				if (j == (dimension - 1))
					ferr_Input << e_input_set[j + i*dimension];
				else
					ferr_Input << e_input_set[j + i*dimension] << " ";
			}
			if (i != (batch_size - 1))
				ferr_Input << std::endl;
		}
		return false;
	}
	return true;
}

//A function for randomMapping. It's not use until now.
float* RandomMapping(const float* h_gaussin,
	const float *h_source,
	const int dimension_after_random_mapping,
	const int dimension_before_random_mapping,
	const int input_set_size)
{
	float *d_source, *d_gaussin, *d_result;

	cublasInit();

	cublasAlloc(input_set_size * dimension_before_random_mapping, sizeof(float), (void**)&d_source);
	cublasAlloc(dimension_after_random_mapping * dimension_before_random_mapping, sizeof(float), (void**)&d_gaussin);
	cublasAlloc(dimension_after_random_mapping * input_set_size, sizeof(float), (void**)&d_result);
	hipblasSetMatrix(dimension_before_random_mapping, input_set_size, sizeof(float), h_source, dimension_before_random_mapping, d_source, dimension_before_random_mapping);
	hipblasSetMatrix(dimension_after_random_mapping, dimension_before_random_mapping, sizeof(float), h_gaussin, dimension_after_random_mapping, d_gaussin, dimension_after_random_mapping);

	hipDeviceSynchronize();

	hipblasSgemm('n', 'n',
		dimension_after_random_mapping, input_set_size, dimension_before_random_mapping,
		1.0f, d_gaussin, dimension_after_random_mapping,
		d_source, dimension_before_random_mapping,
		0.0f, d_result, dimension_after_random_mapping);

	hipDeviceSynchronize();
	hipFree(d_gaussin);
	hipFree(d_source);

	return d_result;
}

unsigned int* SOM(const float* h_inputSet,
	const unsigned int input_set_size,
	const unsigned int dimension,
	const unsigned int height,
	const unsigned int width,
	const unsigned int batch_size,
	const int epochNum,
	const float lambda,
	const float iterNum)
{
	const unsigned int d_input_set_size = input_set_size;							//define the input set size on device
	const unsigned int neuron_number = height * width;								//the number of neuron
	const unsigned int real_dimension = dimension;
	float iter = 0;																	//iteration

	int distance_table_length = (int)(1 +
		neuron_number * (neuron_number - 1) * 0.5);		//the length of distance Table
	float* h_weights = new float[real_dimension * neuron_number];					//weights of each neuron in host memory
	float* h_distance = new float[distance_table_length];							//distance table in host memory
	int* h_position = new int[2 * neuron_number];									//position--(x,y)--of each neuron in host memory

	float* d_weights = 0;															//weights of each neuron in device memory
	float* d_distance = 0;															//distance table in device memory
	float* d_input_set = 0;															//input set in device memory
	unsigned int* d_BID = 0;														//the id of best match neurons in device memory
	float* d_intermediate_result = 0;

	hipMalloc((void**)&d_weights, real_dimension * neuron_number * sizeof(float));
	hipMalloc((void**)&d_distance, distance_table_length * sizeof(float));
	hipMalloc((void**)&d_BID, batch_size * sizeof(unsigned int));
	hipMalloc((void**)&d_intermediate_result, neuron_number * batch_size * sizeof(float));
	hipMalloc((void**)&d_input_set, real_dimension*d_input_set_size*sizeof(float));
	hipMemcpy(d_input_set, h_inputSet, real_dimension*d_input_set_size*sizeof(float), hipMemcpyHostToDevice);

	/*----------------- Initialize the distance table --------------------*/
	bool flag = true;
	int x = 0;
	int y = 0;
	for (int i = 0, t = 0; i < height; ++i)
	{
		x = 0;
		for (int j = 0; j < 2 * width; ++j)
		{

			if (flag)
			{
				h_position[t] = x;
				flag = false;
				++x;
				++t;

			}
			else
			{
				h_position[t] = y;
				flag = true;
				++t;

			}
		}
		y++;
	}

	h_distance[0] = 0;
	for (unsigned int i = 0, t = 1; i < neuron_number - 1; ++i)
	{
		for (unsigned int j = i + 1; j < neuron_number; ++j)
		{
			int dX = (h_position[2 * i] - h_position[2 * j]) * (h_position[2 * i] - h_position[2 * j]);
			int dY = (h_position[2 * i + 1] - h_position[2 * j + 1]) * (h_position[2 * i + 1] - h_position[2 * j + 1]);
			h_distance[t] = dX + dY;
			++t;
		}
	}
	hipMemcpy(d_distance, h_distance, distance_table_length * sizeof(float), hipMemcpyHostToDevice);

	std::cout << "Initialize the positioin done" << std::endl;

	/*-----------Initialize the weights of each neuron---------------------*/
	float *h_temp_weight = new float[neuron_number*real_dimension];
	hipMemcpy(h_temp_weight, d_input_set, neuron_number*real_dimension*sizeof(float), hipMemcpyDeviceToHost);
	for (unsigned int i = 0; i < neuron_number; ++i)
	{
		for (unsigned int j = 0; j < real_dimension; ++j)
		{
			h_weights[i + j * neuron_number] = h_temp_weight[i*real_dimension + j];
		}
	}
	hipMemcpy(d_weights, h_weights, neuron_number* real_dimension  * sizeof(float), hipMemcpyHostToDevice);
	delete[] h_temp_weight;
	h_temp_weight = NULL;
	std::cout << "Initialize the weights done" << std::endl;

	//Let's begin SOM
	for (int i = 0; i < epochNum; i++)
	{
		for (unsigned int iCycle = 0; iCycle < (d_input_set_size / batch_size); iCycle++)
		{
			int inputx = iCycle * batch_size;
			if (!Find_Best_Match_Neuron(d_weights, neuron_number, d_input_set, inputx, batch_size, d_BID, d_intermediate_result))
			{
				break;
			}

			//float sigmaT = (0.5*height * exp(-iter/lambda));
			float sigmaT = 0.28*width*(1 - lambda*iter);
			if (sigmaT < 0.5)
				sigmaT = 0.5;
			std::cout << sigmaT << std::endl;
			sigmaT = 2 * sigmaT * sigmaT;
			if (!Update_Map(d_distance, neuron_number, d_input_set, inputx, d_BID, batch_size, real_dimension, sigmaT, d_weights))
			{
				break;
			}
			iter += iterNum;
		}
	}

	std::cout << "Som training done" << std::endl;

	unsigned int* h_output = new unsigned int[input_set_size];
	for (unsigned int iCycle = 0; iCycle < (d_input_set_size / batch_size); iCycle++)
	{
		int inputx = iCycle * batch_size;
		std::cout << inputx << std::endl;
		if (!Find_Best_Match_Neuron(d_weights, neuron_number, d_input_set, inputx, batch_size, d_BID, d_intermediate_result))
		{
			break;
		}
		hipMemcpy(h_output + inputx, d_BID, batch_size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	/*--------------- check the result of final weights update -----------------*/
	std::ofstream fweightout("../data/somweightsFinal");
	hipMemcpy(h_weights, d_weights, neuron_number * real_dimension * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < neuron_number; ++i)
	{
		for (int j = 0; j < dimension; j++)
		{
			fweightout << h_weights[i + j * neuron_number] << " ";
		}
		fweightout << std::endl;
	}
	fweightout.close();

	std::cout << "everything done" << std::endl;
	hipFree(d_weights);
	hipFree(d_input_set);
	hipFree(d_BID);
	hipFree(d_intermediate_result);
	hipFree(d_distance);
	delete[] h_position;
	delete[] h_weights;
	delete[] h_distance;
	h_distance = NULL;
	h_position = NULL;
	h_weights = NULL;

	return h_output;
}

unsigned int* SOMwithRandomMapping(const float* h_gaussin,
	const float* h_inputSet,
	const unsigned int input_set_size,
	const unsigned int dimension,
	const unsigned int height,
	const unsigned int width,
	const unsigned int batch_size,
	const int epochNum,
	const float lambda,
	const float iterNum)
{
	//std::string logPath = "C:\\Users\\v-zhuche.FAREAST\\Documents\\Visual Studio 2012\\Projects\\myWebApplication\\SOMLog\\";
	std::string logPath = "D:\\SOMLog\\";
	std::ofstream fout(logPath + "log");
	fout << "here we go";
	fout.close();

	const unsigned int d_input_set_size = input_set_size;								//define the input set size on device
	const unsigned int dimension_before_random_mapping = dimension;						//the original dimension of the input set
	const unsigned int dimension_after_random_mapping = DIMENSION;						//dimension after random mapping, can not change
	const unsigned int neuron_number = height * width;									//the number of neuron
	float iter = 0;																		//iteration

	int distance_table_length = (int)(1 +
		neuron_number * (neuron_number - 1) * 0.5);			//the length of distance Table
	float* h_weights = new float[dimension_after_random_mapping * neuron_number];		//weights of each neuron in host memory
	float* h_distance = new float[distance_table_length];								//distance table in host memory
	int* h_position = new int[2 * neuron_number];										//position--(x,y)--of each neuron in host memory

	float* d_weights = 0;																//weights of each neuron in device memory
	float* d_distance = 0;																//distance table in device memory
	unsigned int* d_BID = 0;															//the id of best match neurons in device memory
	float* d_intermediate_result = 0;
	float* d_input_set = 0;																//input set in device memory

	hipMalloc((void**)&d_weights, dimension_after_random_mapping * neuron_number * sizeof(float));
	hipMalloc((void**)&d_distance, distance_table_length * sizeof(float));
	hipMalloc((void**)&d_BID, batch_size * sizeof(unsigned int));
	hipMalloc((void**)&d_intermediate_result, neuron_number * batch_size * sizeof(float));
	d_input_set = RandomMapping(h_gaussin, h_inputSet, dimension_after_random_mapping, dimension_before_random_mapping, input_set_size);

	/*--------------- check the result of random mapping -----------------*/
	/*float* h_checkRM = new float[dimension_after_random_mapping*input_set_size];
	hipMemcpy(h_checkRM,d_input_set,dimension_after_random_mapping*input_set_size*sizeof(float),hipMemcpyDeviceToHost);
	fout.open(logPath+"rmvtrain");
	for(int i =0; i<input_set_size;++i)
	{
	for(int j = 0; j<dimension_after_random_mapping;++j)
	{
	if (j == (dimension_after_random_mapping - 1))
	{
	fout << h_checkRM[j + i*dimension_after_random_mapping];
	}
	else
	{
	fout << h_checkRM[j + i*dimension_after_random_mapping] << " ";
	}

	}
	if(i!=(input_set_size -1 ))
	fout<<std::endl;
	}
	fout.close();
	delete[] h_checkRM;
	h_checkRM = NULL;*/
	//fout.open("../data/vtrain");
	//for(int i =0; i<input_set_size;++i)
	//{
	//	for(int j = 0; j<dimension_before_random_mapping;++j)
	//	{
	//		if(j == (dimension_before_random_mapping - 1))
	//			fout<<h_inputSet[j+i*dimension_before_random_mapping];
	//		else
	//			fout<<h_inputSet[j+i*dimension_before_random_mapping]<<" ";
	//	}
	//	if(i!=(input_set_size -1 ))
	//		fout<<std::endl;
	//}
	//fout.close();

	/*----------------- Initialize the position table --------------------*/
	bool flag = true;
	int x = 0;
	int y = 0;
	for (int i = 0, t = 0; i < height; ++i)
	{
		//x = (i+1)/2;
		x = 0;
		for (int j = 0; j < 2 * width; ++j)
		{
			if (flag)
			{
				h_position[t] = x;
				flag = false;
				++x;
				++t;
			}
			else
			{
				h_position[t] = y;
				flag = true;
				++t;
			}
		}
		y++;
	}
	std::cout << "Initialize the positioin done" << std::endl;
	/*----------------- Initialize the distance table --------------------*/
	h_distance[0] = 0;
	for (unsigned int i = 0, t = 1; i < neuron_number - 1; ++i)
	{
		for (unsigned int j = i + 1; j < neuron_number; ++j)
		{
			int dX = (h_position[2 * i] - h_position[2 * j]) * (h_position[2 * i] - h_position[2 * j]);
			int dY = (h_position[2 * i + 1] - h_position[2 * j + 1]) * (h_position[2 * i + 1] - h_position[2 * j + 1]);

			//if( sgn<int>(dX) == sgn<int>(dY))
			//{
			//	h_distance[t] = abs(dX + dY);
			//}
			//else
			//{
			//	h_distance[t] = abs(dX) > abs(dY) ? abs(dX) : abs(dY);
			//}
			//h_distance[t]  = h_distance[t] *h_distance[t] ;

			h_distance[t] = dX + dY;
			++t;
		}
	}
	hipMemcpy(d_distance, h_distance, distance_table_length * sizeof(float), hipMemcpyHostToDevice);
	std::cout << "Initialize the distance done" << std::endl;

	/*-----------Initialize the weights of each neuron---------------------*/
	hipMemcpy(d_weights, d_input_set, neuron_number* dimension_after_random_mapping  * sizeof(float), hipMemcpyDeviceToDevice);
	std::cout << "Initialize the weights done" << std::endl;

	//Let's begin SOM
	for (int i = 0; i < epochNum; i++)
	{
		for (unsigned int iCycle = 0; iCycle < (d_input_set_size / batch_size); iCycle++)
		{
			int inputx = iCycle * batch_size;
			if (!Find_Best_Match_Neuron(d_weights, neuron_number, d_input_set, inputx, batch_size, d_BID, d_intermediate_result))
			{
				break;
			}

			//float sigmaT = (0.5*height * exp(-iter/lambda));
			float sigmaT = 0.28*width*(1 - lambda*iter);
			if (sigmaT < 1)
				sigmaT = 1;
			std::cout << sigmaT << std::endl;
			sigmaT = 2 * sigmaT * sigmaT;
			if (!Update_Map(d_distance, neuron_number, d_input_set, inputx, d_BID, batch_size, dimension_after_random_mapping, sigmaT, d_weights))
			{
				break;
			}
			iter += iterNum;
		}
	}

	std::cout << "Som training done" << std::endl;

	unsigned int* h_output = new unsigned int[input_set_size];
	for (unsigned int iCycle = 0; iCycle < (d_input_set_size / batch_size); iCycle++)
	{
		int inputx = iCycle * batch_size;
		std::cout << inputx << std::endl;
		if (!Find_Best_Match_Neuron(d_weights, neuron_number, d_input_set, inputx, batch_size, d_BID, d_intermediate_result))
		{
			break;
		}
		hipMemcpy(h_output + inputx, d_BID, batch_size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	/*--------------- check the result of final weights update -----------------*/
	//std::ofstream fweightout("../data/weightsFinal");
	//hipMemcpy(h_weights, d_weights, neuron_number * dimension_after_random_mapping * sizeof(float),hipMemcpyDeviceToHost);
	//for(int i = 0; i < neuron_number; ++i)
	//{
	//	for(int j = 0; j<  dimension_after_random_mapping; j++)
	//	{
	//		fweightout<<h_weights[i + j * neuron_number] <<" ";
	//	}
	//	fweightout<<std::endl;
	//}
	//fweightout.close();

	std::cout << "everything done" << std::endl;
	hipFree(d_weights);
	hipFree(d_input_set);
	hipFree(d_BID);
	hipFree(d_intermediate_result);
	hipFree(d_distance);
	delete[] h_position;
	delete[] h_weights;
	delete[] h_distance;
	h_distance = NULL;
	h_position = NULL;
	h_weights = NULL;

	return h_output;
}

unsigned int* AbstractSOMClassificationwithRandomMapping(std::string mapWeight,
	const float* h_gaussin,
	const float* h_inputSet,
	const unsigned int input_set_size,
	const unsigned int dimension,
	const unsigned int height,
	const unsigned int width,
	const unsigned int batch_size,
	const unsigned int groupsNum)
{
	unsigned int d_input_set_size = input_set_size;										//define the input set size on device
	const unsigned int dimension_before_random_mapping = dimension;						//the original dimension of the input set
	const unsigned int dimension_after_random_mapping = DIMENSION;						//dimension after random mapping, can not change
	const unsigned int neuron_number = height * width;									//the number of neuron

	float* h_weights = new float[dimension_after_random_mapping * neuron_number];		//weights of each neuron in host memory

	float* d_weights = 0;																//weights of each neuron in device memory
	float* d_input_set = 0;																//input set in device memory
	unsigned int* d_BID = 0;															//the id of best match neurons in device memory
	float* d_intermediate_result = 0;
	hipMalloc((void**)&d_weights, dimension_after_random_mapping * neuron_number * sizeof(float));
	hipMalloc((void**)&d_BID, batch_size * sizeof(unsigned int));
	hipMalloc((void**)&d_intermediate_result, neuron_number * batch_size * sizeof(float));

	float* temp_d_input_set = RandomMapping(h_gaussin, h_inputSet, dimension_after_random_mapping - groupsNum, dimension_before_random_mapping, input_set_size);

	if (groupsNum == 0)
	{
		d_input_set = temp_d_input_set;
		temp_d_input_set = NULL;
	}
	else
	{
		float* zero = new float[groupsNum];
		for (int i = 0; i < groupsNum; ++i)
		{
			zero[i] = 0;
		}

		hipMalloc((void**)&d_input_set, dimension_after_random_mapping*d_input_set_size*sizeof(float));
		for (int i = 0; i < d_input_set_size; ++i)
		{
			hipMemcpy(d_input_set + i*dimension_after_random_mapping,
				temp_d_input_set + i*(dimension_after_random_mapping - groupsNum),
				(dimension_after_random_mapping - groupsNum)*sizeof(float),
				hipMemcpyDeviceToDevice);
			hipMemcpy(d_input_set + (i + 1)*(dimension_after_random_mapping)-groupsNum,
				zero,
				groupsNum*sizeof(float),
				hipMemcpyHostToDevice);
		}
		delete[] zero;
		zero = NULL;
		hipFree(temp_d_input_set);
	}
	/* ------------------- check inpout set --------------------*/
	//float* h_checkRM = new float[dimension_after_random_mapping*input_set_size];
	//hipMemcpy(h_checkRM,d_input_set,dimension_after_random_mapping*input_set_size*sizeof(float),hipMemcpyDeviceToHost);
	//std::ofstream fout("../data/rmvtest");
	//for(int i =0; i<input_set_size;++i)
	//{
	//	for(int j = 0; j<dimension_after_random_mapping;++j)
	//	{
	//		if(j == (dimension_after_random_mapping - 1))
	//			fout<<h_checkRM[j+i*dimension_after_random_mapping];
	//		else
	//			fout<<h_checkRM[j+i*dimension_after_random_mapping]<<" ";
	//	}
	//	if(i!=(input_set_size -1 ))
	//		fout<<std::endl;
	//}
	//fout.close();
	//delete[] h_checkRM;
	//h_checkRM = NULL;
	//fout.open("../data/vtest");
	//for(int i =0; i<input_set_size;++i)
	//{
	//	for(int j = 0; j<dimension_before_random_mapping;++j)
	//	{
	//		if(j == (dimension_before_random_mapping - 1))
	//			fout<<h_inputSet[j+i*dimension_before_random_mapping];
	//		else
	//			fout<<h_inputSet[j+i*dimension_before_random_mapping]<<" ";
	//	}
	//	if(i!=(input_set_size -1 ))
	//		fout<<std::endl;
	//}
	//fout.close();

	/*-----------Initialize the weights of each neuron---------------------*/
	std::ifstream fin(mapWeight);
	for (unsigned int i = 0; i < neuron_number; ++i)
	{
		for (unsigned int j = 0; j < dimension_after_random_mapping; ++j)
		{
			fin >> h_weights[i + j * neuron_number];
		}
	}
	hipMemcpy(d_weights, h_weights, neuron_number* dimension_after_random_mapping  * sizeof(float), hipMemcpyHostToDevice);
	std::cout << "Initialize the weights done" << std::endl;

	unsigned int* h_output = new unsigned int[input_set_size];
	for (unsigned int iCycle = 0; iCycle < (d_input_set_size / batch_size); iCycle++)
	{
		int inputx = iCycle * batch_size;
		std::cout << inputx << std::endl;
		if (!output_BID(d_weights, neuron_number, d_input_set, inputx, batch_size, d_BID, d_intermediate_result))
		{
			break;
		}
		hipMemcpy(h_output + inputx, d_BID, batch_size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	std::cout << "everything done" << std::endl;
	hipFree(d_weights);
	hipFree(d_input_set);
	hipFree(d_BID);
	hipFree(d_intermediate_result);
	delete[] h_weights;
	h_weights = NULL;

	return h_output;
}


unsigned int* SOMRefineClassificationwithRandomMapping(const float* h_gaussin,
	const float* h_inputSet,
	const unsigned int input_set_size,
	const unsigned int dimension,
	const unsigned int height,
	const unsigned int width,
	const unsigned int batch_size,
	const unsigned int groupsNum)
{
	return AbstractSOMClassificationwithRandomMapping("../data/somweightsFinal",
		h_gaussin,
		h_inputSet,
		input_set_size,
		dimension,
		height,
		width,
		batch_size,
		groupsNum);
}

unsigned int* SOMClassificationwithRandomMapping(const float* h_gaussin,
	const float* h_inputSet,
	const unsigned int input_set_size,
	const unsigned int dimension,
	const unsigned int height,
	const unsigned int width,
	const unsigned int batch_size,
	const unsigned int groupsNum)
{
	return AbstractSOMClassificationwithRandomMapping("../data/weightsFinal",
		h_gaussin,
		h_inputSet,
		input_set_size,
		dimension,
		height,
		width,
		batch_size,
		groupsNum);
}
